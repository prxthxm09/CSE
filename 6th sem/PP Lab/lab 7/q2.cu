
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10 // Number of elements in input array
#define M 3  // Number of elements in mask array

__global__ void convolution(int *input, int *mask, int *output) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if(tid < N) {
        int result = 0;
        for(int i = 0; i < M; i++) {
            int idx = tid - M/2 + i;
            if(idx >= 0 && idx < N) {
                result += input[idx] * mask[i];
            }
        }
        output[tid] = result;
    }
}

int main() {
    int input[N] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10}; // Input array
    int mask[M] = {1, 0, -1};                       // Mask array
    int output[N];                                  // Resultant array

    int *d_input, *d_mask, *d_output; // Device arrays

    // Allocate memory on device
    hipMalloc(&d_input, N * sizeof(int));
    hipMalloc(&d_mask, M * sizeof(int));
    hipMalloc(&d_output, N * sizeof(int));

    // Copy input and mask arrays from host to device
    hipMemcpy(d_input, input, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, M * sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernel
    convolution<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_mask, d_output);

    // Copy result back to host
    hipMemcpy(output, d_output, N * sizeof(int), hipMemcpyDeviceToHost);

    // Print result
    printf("Result: ");
    for(int i = 0; i < N; i++) {
        printf("%d ", output[i]);
    }
    printf("\n");

    // Free device memory
    hipFree(d_input);
    hipFree(d_mask);
    hipFree(d_output);

    return 0;
}


#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10 // Size of vectors

__global__ void axpy(float alpha, float *x, float *y) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if(tid < N) {
        y[tid] = alpha * x[tid] + y[tid];
    }
}

int main() {
    float alpha = 2.0;       // Scalar value
    float x[N], y[N];        // Input and output vectors

    // Initialize input vectors
    for(int i = 0; i < N; i++) {
        x[i] = i + 1;         // Example values
        y[i] = 10 - i;        // Example values
    }

    float *d_x, *d_y;        // Device arrays

    // Allocate memory on device
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    // Copy input vectors from host to device
    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernel
    axpy<<<blocksPerGrid, threadsPerBlock>>>(alpha, d_x, d_y);

    // Copy result back to host
    hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    // Print result
    printf("Result: ");
    for(int i = 0; i < N; i++) {
        printf("%.2f ", y[i]);
    }
    printf("\n");

    // Free device memory
    hipFree(d_x);
    hipFree(d_y);

    return 0;
}

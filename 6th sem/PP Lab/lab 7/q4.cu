
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define N 10 // Size of array

__global__ void computeSine(float *angles, float *sine) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if(tid < N) {
        sine[tid] = sinf(angles[tid]);
    }
}

int main() {
    float angles[N]; // Input array containing angles in radians
    float sine[N];   // Output array containing sine values

    // Initialize input array with example angles
    for(int i = 0; i < N; i++) {
        angles[i] = (i + 1) * 0.1; // Example values
    }

    float *d_angles, *d_sine; // Device arrays

    // Allocate memory on device
    hipMalloc(&d_angles, N * sizeof(float));
    hipMalloc(&d_sine, N * sizeof(float));

    // Copy input array from host to device
    hipMemcpy(d_angles, angles, N * sizeof(float), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernel
    computeSine<<<blocksPerGrid, threadsPerBlock>>>(d_angles, d_sine);

    // Copy result back to host
    hipMemcpy(sine, d_sine, N * sizeof(float), hipMemcpyDeviceToHost);

    // Print result
    printf("Sine values:\n");
    for(int i = 0; i < N; i++) {
        printf("sin(%.2f) = %.6f\n", angles[i], sine[i]);
    }

    // Free device memory
    hipFree(d_angles);
    hipFree(d_sine);

    return 0;
}
